#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "material_parameters.h"
#include "state.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

// -----------------------------------------------------------------------------

RT_FUNCTION float fresnel2( float cos_theta_i, float cos_theta_t, float eta )
{
    const float rs = ( cos_theta_i - cos_theta_t*eta ) / 
                     ( cos_theta_i + eta*cos_theta_t );
    const float rp = ( cos_theta_i*eta - cos_theta_t ) /
                     ( cos_theta_i*eta + cos_theta_t );

    return 0.5f * ( rs*rs + rp*rp );
}

RT_FUNCTION float3 logf2( float3 v )
{
    return make_float3( logf(v.x), logf(v.y), logf(v.z) );
}

RT_CALLABLE_PROGRAM void Pdf(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	prd.pdf = 1.0f;
}

RT_CALLABLE_PROGRAM void Sample(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	const float3 w_out = -ray.direction;
	float3 normal = state.normal;
	float cos_theta_i = optix::dot( w_out, normal );

	float eta;
	float3 transmittance = make_float3(1.0f);
	float3 extinction = -logf(make_float3(0.905f, 0.63f, 0.3));
	if( cos_theta_i > 0.0f )
	{
		eta = 1.45f;
	} 
	else
	{
		transmittance = optix::expf(-extinction * t_hit);
		eta = 1.0f / 1.45f;
		cos_theta_i = -cos_theta_i;
		normal = -normal;
	}
	//intData.mat.color = transmittance;

	float3 w_t;
	const bool tir  = !optix::refract( w_t, -w_out, normal, eta );
	const float cos_theta_t = -optix::dot( normal, w_t );
	const float R  = tir  ? 1.0f : fresnel( cos_theta_i, cos_theta_t, eta );

	const float z = rnd(prd.seed);
	if( z <= R )
	{
		// Reflect
		prd.origin = state.fhp;
		prd.direction =  optix::reflect( -w_out, normal );
	}
	else
	{
		// Refract
		prd.origin = state.bhp;
		prd.direction = w_t;
	} 
	//prd.origin = state.bhp;
	//prd.direction = -w_out;
}

RT_CALLABLE_PROGRAM float3 Eval(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	return mat.color;
}

