#include "hip/hip_runtime.h"
/*
 Copyright Disney Enterprises, Inc.  All rights reserved.

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License
 and the following modification to it: Section 6 Trademarks.
 deleted and replaced with:

 6. Trademarks. This License does not grant permission to use the
 trade names, trademarks, service marks, or product names of the
 Licensor and its affiliates, except as required for reproducing
 the content of the NOTICE file.

 You may obtain a copy of the License at
 http://www.apache.org/licenses/LICENSE-2.0
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "material_parameters.h"
#include "state.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );


RT_CALLABLE_PROGRAM void Pdf(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	float3 n = state.ffnormal;
	float3 L = prd.direction;
	
	float pdfDiff = abs(dot(L, n))* (1.0f / M_PIf);

	prd.pdf =  pdfDiff;

}

RT_CALLABLE_PROGRAM void Sample(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	float3 N = state.ffnormal;
	prd.origin = state.fhp;

	float3 dir;
	
	float r1 = rnd(prd.seed);
	float r2 = rnd(prd.seed);

	optix::Onb onb( N );

	cosine_sample_hemisphere(r1, r2, dir);
	onb.inverse_transform(dir);
	
	prd.direction = dir;
}


RT_CALLABLE_PROGRAM float3 Eval(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	float3 N = state.ffnormal;
	float3 V = -ray.direction;
	float3 L = prd.direction;

	float NDotL = dot(N, L);
	float NDotV = dot(N, V);
	if (NDotL <= 0.0f || NDotV <= 0.0f) return make_float3(0.0f);

	float3 out = (1.0f / M_PIf) * mat.color;

	return out * clamp(dot(N, L), 0.0f, 1.0f);
}
