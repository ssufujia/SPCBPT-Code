#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"

using namespace optix;

#define PI 3.14159265358979323846f

rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable( float3, texcoord, attribute texcoord, );

rtDeclareVariable(Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(float3, var_color, , );
rtDeclareVariable(float, var_metallic, , );
rtDeclareVariable(float, var_subsurface, , );
rtDeclareVariable(float, var_specular, , );
rtDeclareVariable(float, var_roughness, , );
rtDeclareVariable(float, var_specularTint, , );
rtDeclareVariable(float, var_anisotropic, , );
rtDeclareVariable(float, var_sheen, , );
rtDeclareVariable(float, var_sheenTint, , );
rtDeclareVariable(float, var_clearcoat, , );
rtDeclareVariable(float, var_clearcoatGloss, , );

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable( float2, Kd_map_scale, , );

struct Material
{
	__device__ __inline__ Material()
	{
		color = var_color;
		metallic = var_metallic;
		subsurface = var_subsurface;
		specular = var_specular;
		roughness = var_roughness;
		specularTint = var_specularTint;
		anisotropic = var_anisotropic;
		sheen = var_sheen;
		sheenTint = var_sheenTint;
		clearcoat = var_clearcoat;
		clearcoatGloss = var_clearcoatGloss;
	}

	float3 color;
	float metallic;
	float subsurface;
	float specular;
	float roughness;
	float specularTint;
	float anisotropic;
	float sheen;
	float sheenTint;
	float clearcoat;
	float clearcoatGloss;
};


static __device__ __inline__ float sqr(float x) { return x*x; }

static __device__ __inline__ float SchlickFresnel(float u)
{
    float m = clamp(1.0f-u, 0.0f, 1.0f);
    float m2 = m*m;
    return m2*m2*m; // pow(m,5)
}

static __device__ __inline__ float GTR1(float NDotH, float a)
{
    if (a >= 1.0f) return (1.0f/PI);
    float a2 = a*a;
    float t = 1.0f + (a2-1.0f)*NDotH*NDotH;
    return (a2-1.0f) / (PI*logf(a2)*t);
}

static __device__ __inline__ float GTR2(float NDotH, float a)
{
    float a2 = a*a;
    float t = 1.0f + (a2-1.0f)*NDotH*NDotH;
    return a2 / (PI * t*t);
}

static __device__ __inline__ float GTR2_aniso(float NDotH, float HDotX, float HDotY, float ax, float ay)
{
    return 1.0f / ( PI * ax*ay * sqr( sqr(HDotX/ax) + sqr(HDotY/ay) + NDotH*NDotH ));
}

static __device__ __inline__ float smithG_GGX(float NDotv, float alphaG)
{
    float a = alphaG*alphaG;
    float b = NDotv*NDotv;
    return 1.0f/(NDotv + sqrtf(a + b - a*b));
}

static __device__ __inline__ float BRDFPdf(const Material &mat, const float3& P, const float3& n, const float3& V, const float3& L)
{
    const float a = max(0.001f, mat.roughness);
	float b = lerp(0.1f, 0.001f, mat.clearcoatGloss);
	float ratio = 1.0f / (1.0f + mat.clearcoat);
	float diffuseRatio = 0.5f * (1.f - mat.metallic);
	float specularRatio = 1.f - diffuseRatio;

	const float3 half = normalize(L+V);

	const float cosThetaHalf = abs(dot(half, n));
    const float pdfGTR2 = GTR2(cosThetaHalf, a) * cosThetaHalf;
	const float pdfGTR1 = GTR1(cosThetaHalf, b) * cosThetaHalf;

	// calculate pdf for each method given outgoing light vector
	float pdfSpec = lerp(pdfGTR1, pdfGTR2, ratio) / (4.0 * abs(dot(L, half)));

    float pdfDiff = abs(dot(L, n))* (1.0f / PI);

    // weight pdfs according to roughness
	//return lerp(pdfSpec, pdfDiff, mat.roughness);
	return diffuseRatio * pdfDiff + specularRatio * pdfSpec;

}

// generate an importance sampled brdf direction
static __device__ __inline__ float3 BRDFSample(const Material &mat, const float3& P, const float3& V, const float3&n)
{
    float3 light;
	
    const float select = rnd(prd_radiance.seed);

	const float r1 = rnd(prd_radiance.seed);
	const float r2 = rnd(prd_radiance.seed);
	const optix::Onb onb( n );

	float diffuseRatio = 0.5f * (1.f - mat.metallic);

	if (select < diffuseRatio) //roughness
    {
        // sample diffuse
		cosine_sample_hemisphere( r1, r2, light );
		onb.inverse_transform(light);
    }
    else
    {
		const float a = max(0.001f, mat.roughness);

        const float phiHalf = r1 * 2.0f * PI;
        
        const float cosThetaHalf = sqrtf((1.0f-r2)/(1.0f + (sqr(a)-1.0f)*r2));      
        const float sinThetaHalf = sqrtf(max(0.0f,1.0f-sqr(cosThetaHalf)));
        const float sinPhiHalf = sinf(phiHalf);
        const float cosPhiHalf = cosf(phiHalf);

		float3 half = make_float3(sinThetaHalf*sinPhiHalf, sinThetaHalf*cosPhiHalf, cosThetaHalf);
		onb.inverse_transform(half);

        light = 2.0f*dot(V, half)*half - V;

    }
	return light;
}


static __device__ __inline__ float3 BRDFEval(const Material &mat, const float3& P, const float3& N, const float3& V, const float3& L)
{
	float NDotL = dot(N, L);
	float NDotV = dot(N, V);
	if (NDotL <= 0.0f || NDotV <= 0.0f) 
		return make_float3(0.0f);

	float3 H = normalize(L + V);
	float NDotH = dot(N, H);
	float LDotH = dot(L, H);

	float3 Cdlin = mat.color;
	float Cdlum = 0.3f*Cdlin.x + 0.6f*Cdlin.y + 0.1f*Cdlin.z; // luminance approx.

	float3 Ctint = Cdlum > 0.0f ? Cdlin / Cdlum : make_float3(1.0f); // normalize lum. to isolate hue+sat
	float3 Cspec0 = lerp(mat.specular*0.08f*lerp(make_float3(1.0f), Ctint, mat.specularTint), Cdlin, mat.metallic);
	float3 Csheen = lerp(make_float3(1.0f), Ctint, mat.sheenTint);

	// Diffuse fresnel - go from 1 at normal incidence to .5 at grazing
	// and mix in diffuse retro-reflection based on roughness
	float FL = SchlickFresnel(NDotL), FV = SchlickFresnel(NDotV);
	float Fd90 = 0.5f + 2.0f * LDotH*LDotH * mat.roughness;
	float Fd = lerp(1.0f, Fd90, FL) * lerp(1.0f, Fd90, FV);

	// Based on Hanrahan-Krueger brdf approximation of isotrokPic bssrdf
	// 1.25 scale is used to (roughly) preserve albedo
	// Fss90 used to "flatten" retroreflection based on roughness
	float Fss90 = LDotH*LDotH*mat.roughness;
	float Fss = lerp(1.0f, Fss90, FL) * lerp(1.0f, Fss90, FV);
	float ss = 1.25f * (Fss * (1.0f / (NDotL + NDotV) - 0.5f) + 0.5f);

	// specular
	//float aspect = sqrt(1-mat.anisotrokPic*.9);
	//float ax = Max(.001f, sqr(mat.roughness)/aspect);
	//float ay = Max(.001f, sqr(mat.roughness)*aspect);
	//float Ds = GTR2_aniso(NDotH, Dot(H, X), Dot(H, Y), ax, ay);
	float a = max(0.001f, mat.roughness);
	float Ds = GTR2(NDotH, a);
	float FH = SchlickFresnel(LDotH);
	float3 Fs = lerp(Cspec0, make_float3(1.0f), FH);
	float roughg = sqr(mat.roughness*0.5f + 0.5f);
	float Gs = smithG_GGX(NDotL, roughg) * smithG_GGX(NDotV, roughg);

	// sheen
	float3 Fsheen = FH * mat.sheen * Csheen;

	// clearcoat (ior = 1.5 -> F0 = 0.04)
	float Dr = GTR1(NDotH, lerp(0.1f, 0.001f, mat.clearcoatGloss));
	float Fr = lerp(0.04f, 1.0f, FH);
	float Gr = smithG_GGX(NDotL, 0.25f) * smithG_GGX(NDotV, 0.25f);

	float3 out = ((1.0f / PI) * lerp(Fd, ss, mat.subsurface)*Cdlin + Fsheen)
		* (1.0f - mat.metallic)
		+ Gs*Fs*Ds + 0.25f*mat.clearcoat*Gr*Fr*Dr;

    return out;
}

RT_PROGRAM void closest_hit_radiance()
{
	Material mat;
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 brdfDir = BRDFSample(mat, front_hit_point, -ray.direction, ffnormal);

	float brdfPdf = BRDFPdf(mat, front_hit_point, ffnormal, -ray.direction, brdfDir);

	// reflectance
	float3 f = BRDFEval(mat, front_hit_point, ffnormal, -ray.direction, brdfDir);

	// update throughput with primitive reflectance
	if (brdfPdf > 0.0f)
	{
		prd_radiance.reflectance *= f * clamp(dot(ffnormal,brdfDir), 0.0f, 1.0f) / brdfPdf;
		// update path direction
		prd_radiance.origin = front_hit_point;
		prd_radiance.direction = brdfDir;	
	}
	//else
	//	prd_radiance.done = true;
}
