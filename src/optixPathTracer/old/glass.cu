#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "prd.h"
#include "random.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );

// Transmittance at unit distance ( a color with each channel in [0,1] range)
rtDeclareVariable(float3,       unit_transmittance, , );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// -----------------------------------------------------------------------------

static __device__ __inline__ float fresnel( float cos_theta_i, float cos_theta_t, float eta )
{
    const float rs = ( cos_theta_i - cos_theta_t*eta ) / 
                     ( cos_theta_i + eta*cos_theta_t );
    const float rp = ( cos_theta_i*eta - cos_theta_t ) /
                     ( cos_theta_i*eta + cos_theta_t );

    return 0.5f * ( rs*rs + rp*rp );
}

static __device__ __inline__ float3 logf( float3 v )
{
    return make_float3( logf(v.x), logf(v.y), logf(v.z) );
}

// -----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_radiance()
{
    const float3 w_out = -ray.direction;
    float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float cos_theta_i = optix::dot( w_out, normal );

    float eta;
    float3 transmittance = make_float3( 1.0f );
    if( cos_theta_i > 0.0f ) {
        // Ray is entering 
        eta = refraction_index;  // Note: does not handle nested dielectrics
    } else {
        // Ray is exiting; apply Beer's Law.
        // This is derived in Shirley's Fundamentals of Graphics book.
        // The "unit_transmittance" is transmittance at unit distance and must
        // be between 0 and 1, so that log(...) is negative.
        transmittance = optix::expf( logf(unit_transmittance) * t_hit );
        eta         = 1.0f / refraction_index;
        cos_theta_i = -cos_theta_i;
        normal      = -normal;
    }

    float3 w_t;
    const bool tir           = !optix::refract( w_t, -w_out, normal, eta );

    const float cos_theta_t  = -optix::dot( normal, w_t );
    const float R            = tir  ?
                               1.0f :
                               fresnel( cos_theta_i, cos_theta_t, eta );

    // Importance sample the Fresnel term
    const float z = rnd( prd_radiance.seed );
    if( z <= R ) {
        // Reflect
        const float3 w_in = optix::reflect( -w_out, normal ); 
        const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
        prd_radiance.origin = fhp;
        prd_radiance.direction = w_in; 
        prd_radiance.reflectance *= reflection_color*transmittance;
    } else {
        // Refract
        const float3 w_in = w_t;
        const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);
        prd_radiance.origin = bhp;
        prd_radiance.direction = w_in; 
        prd_radiance.reflectance *= refraction_color*transmittance;
    }

    // Note: we do not trace the ray for the next bounce here, we just set it up for
    // the ray-gen program using per-ray data. 

}


