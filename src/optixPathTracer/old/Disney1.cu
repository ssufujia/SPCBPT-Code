#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"

using namespace optix;

#define PI 3.14159265358979323846f
#define DENOM_EPS 1e-8f
#define ROUGHNESS_EPS 0.0001f
#define WHITE make_float3(1.f, 1.f, 1.f)

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable(float2, Kd_map_scale, , );

struct Material
{
	__device__ __inline__ Material()
	{
		base_color = make_float3(0.6f, 0.035f, 0.024f);
		metallic = 1.0f;
		subsurface = 0.0f;
		specular = 0.0f;
		roughness = 0.0f;
		specular_tint = 0.0f;
		anisotropy = 0.0f;
		sheen = 0.0f;
		sheen_tint = 0.0f;
		clearcoat = 0.0f;
		clearcoat_gloss = 0.0f;
	}

	float3 base_color;
	float metallic;
	float subsurface;
	float specular;
	float roughness;
	float specular_tint;
	float anisotropy;
	float sheen;
	float sheen_tint;
	float clearcoat;
	float clearcoat_gloss;
};

static __device__ __inline__ float SchlickFresnelReflectance(float u)
{
	float m = clamp(1.f - u, 0.f, 1.f);
	float m2 = m * m;
	return m2 * m2 * m;
}

static __device__ __inline__ float GTR1(float ndoth, float a)
{
	if (a >= 1.f) return 1.f / PI;

	float a2 = a * a;
	float t = 1.f + (a2 - 1.f) * ndoth * ndoth;
	return (a2 - 1.f) / (PI * log(a2) * t);
}

static __device__ __inline__ float GTR2_Aniso(float ndoth, float hdotx, float hdoty, float ax, float ay)
{
	float hdotxa2 = (hdotx / ax);
	hdotxa2 *= hdotxa2;
	float hdotya2 = (hdoty / ay);
	hdotya2 *= hdotya2;
	float denom = hdotxa2 + hdotya2 + ndoth * ndoth;
	return denom > 1e-5 ? (1.f / (PI * ax * ay * denom * denom)) : 0.f;
}


static __device__ __inline__ float SmithGGX_G(float ndotv, float a)
{
	float a2 = a * a;
	float b = ndotv * ndotv;
	return 1.f / (ndotv + sqrtf(a2 + b - a2 * b));
}

static __device__ __inline__ float SmithGGX_G_Aniso(float ndotv, float vdotx, float vdoty, float ax, float ay)
{
	float vdotxax2 = (vdotx * ax) * (vdotx * ax);
	float vdotyay2 = (vdoty * ay) * (vdoty * ay);
	return 1.f / (ndotv + sqrtf(vdotxax2 + vdotyay2 + ndotv * ndotv));
}


static __device__ __inline__ float Disney_GetPdf(Material &mat, float3 wi, float3 wo)
{
	float aspect = sqrtf(1.f - mat.anisotropy * 0.9f);

	float ax = max(0.001f, mat.roughness * mat.roughness * (1.f + mat.anisotropy));
	float ay = max(0.001f, mat.roughness * mat.roughness * (1.f - mat.anisotropy));
	float3 wh = normalize(wo + wi);
	float ndotwh = fabs(wh.y);
	float hdotwo = fabs(dot(wh, wo));

	float d_pdf = fabs(wo.y) / PI;
	float r_pdf = GTR2_Aniso(ndotwh, wh.x, wh.z, ax, ay) * ndotwh / (4.f * hdotwo);
	float c_pdf = GTR1(ndotwh, lerp(0.1f, 0.001f, mat.clearcoat_gloss)) * ndotwh / (4.f * hdotwo);

	float3 cd_lin = mat.base_color;//make_float3(powf(mat.base_color.x, 2.2f), powf(mat.base_color.y, 2.2f), powf(mat.base_color.z, 2.2f));
	// Luminance approximmation
	float cd_lum = dot(cd_lin, make_float3(0.3f, 0.6f, 0.1f));

	// Normalize lum. to isolate hue+sat
	float3 c_tint = cd_lum > 0.f ? (cd_lin / cd_lum) : WHITE;

	float3 c_spec0 = lerp(mat.specular * 0.1f * lerp(WHITE,
		c_tint, mat.specular_tint),
		cd_lin, mat.metallic);

	float cs_lum = dot(c_spec0, make_float3(0.3f, 0.6f, 0.1f));

	float cs_w = cs_lum / (cs_lum + (1.f - mat.metallic) * cd_lum);

	return c_pdf * mat.clearcoat + (1.f - mat.clearcoat) * (cs_w * r_pdf + (1.f - cs_w) * d_pdf);
}


static __device__ __inline__ float3 Disney_Evaluate(Material &mat, float3 wi, float3 wo)
{

	float ndotwi = fabs(wi.y);
	float ndotwo = fabs(wo.y);

	float3 h = normalize(wi + wo);
	float ndoth = fabs(h.y);
	float hdotwo = fabs(dot(h, wo));

	float3 cd_lin = mat.base_color;//make_float3(powf(mat.base_color.x, 2.2f), powf(mat.base_color.y, 2.2f), powf(mat.base_color.z, 2.2f));
	// Luminance approximmation
	float cd_lum = dot(cd_lin, make_float3(0.3f, 0.6f, 0.1f));

	// Normalize lum. to isolate hue+sat
	float3 c_tint = cd_lum > 0.f ? (cd_lin / cd_lum) : WHITE;

	float3 c_spec0 = lerp(mat.specular * 0.1f * lerp(WHITE,
		c_tint, mat.specular_tint),
		cd_lin, mat.metallic);

	float3 c_sheen = lerp(WHITE, c_tint, mat.sheen_tint);

	// Diffuse fresnel - go from 1 at normal incidence to 0.5 at grazing
	// and lerp in diffuse retro-reflection based on mat.roughness
	float f_wo = SchlickFresnelReflectance(ndotwo);
	float f_wi = SchlickFresnelReflectance(ndotwi);

	float fd90 = 0.5f + 2 * hdotwo * hdotwo * mat.roughness;
	float fd = lerp(1.f, fd90, f_wo) * lerp(1.f, fd90, f_wi);

	// Based on Hanrahan-Krueger brdf approximation of isotropic bssrdf
	// 1.25 scale is used to (roughly) preserve albedo
	// fss90 used to "flatten" retroreflection based on mat.roughness
	float fss90 = hdotwo * hdotwo * mat.roughness;
	float fss = lerp(1.f, fss90, f_wo) * lerp(1.f, fss90, f_wi);
	float ss = 1.25f * (fss * (1.f / (ndotwo + ndotwi) - 0.5f) + 0.5f);

	// mat.specular
	float ax = max(0.001f, mat.roughness * mat.roughness * (1.f + mat.anisotropy));
	float ay = max(0.001f, mat.roughness * mat.roughness * (1.f - mat.anisotropy));
	float ds = GTR2_Aniso(ndoth, h.x, h.z, ax, ay);
	float fh = SchlickFresnelReflectance(hdotwo);
	float3 fs = lerp(c_spec0, WHITE, fh);

	float gs;
	gs = SmithGGX_G_Aniso(ndotwo, wo.x, wo.z, ax, ay);
	gs *= SmithGGX_G_Aniso(ndotwi, wi.x, wi.z, ax, ay);

	// mat.sheen
	float3 f_sheen = fh * mat.sheen * c_sheen;

	// mat.clearcoat (ior = 1.5 -> F0 = 0.04)
	float dr = GTR1(ndoth, lerp(0.1f, 0.001f, mat.clearcoat_gloss));
	float fr = lerp(0.04f, 1.f, fh);
	float gr = SmithGGX_G(ndotwo, 0.25f) * SmithGGX_G(ndotwi, 0.25f);

	return ((1.f / PI) * lerp(fd, ss, mat.subsurface) * cd_lin + f_sheen) *
		(1.f - mat.metallic) + gs * fs * ds + mat.clearcoat * gr * fr * dr;
}


static __device__ __inline__ float3 GetOrthoVector(float3 n)
{
	float3 p;

	if (fabs(n.z) > 0.f) {
		float k = sqrt(n.y*n.y + n.z*n.z);
		p.x = 0; p.y = -n.z / k; p.z = n.y / k;
	}
	else {
		float k = sqrt(n.x*n.x + n.y*n.y);
		p.x = n.y / k; p.y = -n.x / k; p.z = 0;
	}

	return normalize(p);
}


static __device__ __inline__ float3 Sample_MapToHemisphere(float3 n, float e)
{
	// Construct basis
	float2 sample = make_float2(rnd(prd_radiance.seed), rnd(prd_radiance.seed));
	float3 u = GetOrthoVector(n);
	float3 v = cross(u, n);
	u = cross(n, v);

	// Calculate 2D sample
	float r1 = sample.x;
	float r2 = sample.y;

	// Transform to spherical coordinates
	float sinpsi = sin(2 * PI*r1);
	float cospsi = cos(2 * PI*r1);
	float costheta = pow(1.f - r2, 1.f / (e + 1.f));
	float sintheta = sqrt(1.f - costheta * costheta);

	// Return the result
	return normalize(u * sintheta * cospsi + v * sintheta * sinpsi + n * costheta);
}

static __device__ __inline__ float3 Disney_Sample(float3 normal, Material &mat, float3 wi, float3* wo, float* pdf)
{
	float2 sample = make_float2(rnd(prd_radiance.seed), rnd(prd_radiance.seed));
	float ax = max(0.001f, mat.roughness * mat.roughness * (1.f + mat.anisotropy));
	float ay = max(0.001f, mat.roughness * mat.roughness * (1.f - mat.anisotropy));

	float3 wh;


	if (sample.x < mat.clearcoat)
	{
		sample.x /= (mat.clearcoat);

		float a = lerp(0.1f, 0.001f, mat.clearcoat_gloss);
		float ndotwh = sqrtf((1.f - powf(a*a, 1.f - sample.y)) / (1.f - a*a));
		float sintheta = sqrtf(1.f - ndotwh * ndotwh);
		wh = normalize(make_float3(cos(2.f * PI * sample.x) * sintheta,
			ndotwh,
			sin(2.f * PI * sample.x) * sintheta));

		*wo = -wi + 2.f*fabs(dot(wi, wh)) * wh;

	}
	else
	{
		sample.x -= (mat.clearcoat);
		sample.x /= (1.f - mat.clearcoat);

		float3 cd_lin = mat.base_color;//make_float3(powf(mat.base_color.x, 2.2f), powf(mat.base_color.y, 2.2f), powf(mat.base_color.z, 2.2f));
		// Luminance approximmation
		float cd_lum = dot(cd_lin, make_float3(0.3f, 0.6f, 0.1f));

		// Normalize lum. to isolate hue+sat
		float3 c_tint = cd_lum > 0.f ? (cd_lin / cd_lum) : WHITE;

		float3 c_spec0 = lerp(mat.specular * 0.3f * lerp(WHITE,
			c_tint, mat.specular_tint),
			cd_lin, mat.metallic);

		float cs_lum = dot(c_spec0, make_float3(0.3f, 0.6f, 0.1f));

		float cs_w = cs_lum / (cs_lum + (1.f - mat.metallic) * cd_lum);

		if (sample.y < cs_w)
		{
			sample.y /= cs_w;

			float t = sqrtf(sample.y / (1.f - sample.y));
			wh = normalize(make_float3(t * ax * cos(2.f * PI * sample.x),
				1.f,
				t * ay * sin(2.f * PI * sample.x)));

			*wo = -wi + 2.f*fabs(dot(wi, wh)) * wh;
		}
		else
		{
			sample.y -= cs_w;
			sample.y /= (1.f - cs_w);

			*wo = Sample_MapToHemisphere(make_float3(0.0f,1.0f,0.0f), 1.f);

			wh = normalize(*wo + wi);
		}
	}

	//float ndotwh = fabs(wh.y);
	//float hdotwo = fabs(dot(wh, *wo));

	*pdf = Disney_GetPdf(mat, wi, *wo);

	return Disney_Evaluate(mat, wi, *wo);
}

RT_PROGRAM void closest_hit_radiance()
{
	Material mat;
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	const float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 out;
	float pdf;
	const optix::Onb onb(ffnormal);

	float3 f = Disney_Sample(ffnormal, mat, -ray.direction, &out, &pdf);
	//onb.inverse_transform(out);

	// update throughput with primitive reflectance
	if (pdf > 0.0f)
	{
		prd_radiance.reflectance *= f * clamp(dot(ffnormal, out), 0.0f, 1.0f) / pdf;
		// update path direction
		prd_radiance.origin = front_hit_point;
		prd_radiance.direction = out;
	}
	else
		prd_radiance.done = true;
}
