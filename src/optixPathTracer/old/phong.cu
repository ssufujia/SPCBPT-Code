#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"

using namespace optix;

#define PI 3.14159265358979323846f

rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable( float3, texcoord, attribute texcoord, );

rtDeclareVariable(Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable( float2, Kd_map_scale, , );

struct Material
{
	__device__ __inline__ Material()
	{
		color = make_float3(0.6f,0.3f,0.2f);
		kd = 0.1f;
		ks = 0.9f;
		shininess = 512.0f;
	}

	float3 color;
	float kd;
	float ks;
	float shininess;
};

static __device__ __inline__ float3 BRDFEval(const Material &mat, const float3& P, const float3& N, const float3& V, float3& newdir)
{
	float theta = 0.0f;
	float phi = 0.0f;
	float brdf = 0.0f;
	float pdf = 0.0f;
	float3 reflectionDir = reflect(V, N);
	float u = rnd(prd_radiance.seed);

	// diffuse sample
	if (u < mat.kd) {
		phi = 2.0f * M_PI * rnd(prd_radiance.seed);
		theta = acos(sqrt(rnd(prd_radiance.seed)));
		pdf = (1.0f / M_PI) * cos(theta);
		brdf += (1.0f / M_PI) * mat.kd;
	}

	// specular sample - check the reference to understand where these calculations are derived
	if ((u >= mat.kd) && (u < mat.kd + mat.ks)) {

		theta = acos(pow(rnd(prd_radiance.seed), 1 / (mat.shininess + 1) ));
		if (theta > M_PI * 0.5) 
			theta = M_PI * 0.5;
		phi = 2.0f * M_PI * rnd(prd_radiance.seed);

		pdf = ((mat.shininess + 1) / (2.0f * M_PI)) * pow(cos(theta), mat.shininess);
		brdf = ((mat.shininess + 2) / (2.0f * M_PI)) * pow(cos(theta), mat.shininess) * mat.ks;
	}

	if (u > (mat.ks + mat.kd)) {
		return make_float3(0, 0, 0);
	}

	float3 scy;
	if (u < mat.kd) 
		scy = N;	
	else		
		scy = reflectionDir;

	float3 upv = make_float3(0, 1, 0);
	if (dot(scy, upv) > 0.99f) {
		upv = make_float3(1, 0, 0);
	}

	float3 scx = normalize(cross(upv, scy));
	float3 scz = normalize(cross(scx, scy));

	float ssx = cos(phi) * sin(theta);
	float ssy = cos(theta);
	float ssz = sin(phi) * sin(theta);

	// construct transformation matrix

	Matrix3x3 rotmatrix;
	rotmatrix.setCol(0, scx);
	rotmatrix.setCol(1, scy);
	rotmatrix.setCol(2, scz);

	newdir = rotmatrix * make_float3(ssx, ssy, ssz);

	/*if (dot(newdir, N) < 0) {
		newdir = N;
	}*/

	if (isnan(newdir.x) || isnan(newdir.y) || isnan(newdir.z)) {
		newdir = make_float3(0, 1, 0);
	}

	if (pdf <= 0.0f || brdf <= 0.0f) {
		pdf = 1.0f;
		brdf = 1.0f;
	}

	if (pdf < 0.0001f) {
		pdf = 0.0001f;
	}

	newdir = normalize(newdir);

	float3 mask = (mat.color * dot(N, newdir) * brdf) / pdf;

    return mask;
}

RT_PROGRAM void closest_hit_radiance()
{
	Material mat;
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	// reflectance
	float3 newdir;

	// update throughput with primitive reflectance
	prd_radiance.reflectance *= BRDFEval(mat, front_hit_point, ffnormal, ray.direction, newdir);
	prd_radiance.origin = front_hit_point;
	prd_radiance.direction = newdir;
}
