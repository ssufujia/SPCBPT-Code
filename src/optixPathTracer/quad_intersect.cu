#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_world.h>
#include "intersection_refinement.h"

using namespace optix;

rtDeclareVariable(float4, plane, , );
rtDeclareVariable(float3, v1, , );
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, anchor, , );

rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float4, geometry_color, attribute geometry_color, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx)
{
	float3 n = make_float3(plane);
	float dt = dot(ray.direction, n);
	float t = (plane.w - dot(n, ray.origin)) / dt;
	if (t > ray.tmin && t < ray.tmax) {
		float3 p = ray.origin + ray.direction * t;
		float3 vi = p - anchor;
		float a1 = dot(v1, vi);
		if (a1 >= 0 && a1 <= 1) {
			float a2 = dot(v2, vi);
			if (a2 >= 0 && a2 <= 1) {
				if (rtPotentialIntersection(t)) {
					shading_normal = geometric_normal = n;
					texcoord = make_float3(a1, a2, 0);
					geometry_color = make_float4(1.0f);

					refine_and_offset_hitpoint(ray.origin + t * ray.direction, ray.direction,
						n, anchor,
						back_hit_point, front_hit_point);

					rtReportIntersection(0);
				}
			}
		}
	}
}

RT_PROGRAM void intersect2(int primIdx)
{
	float3 n = make_float3(plane);	
	float3 tv1 = v1 / dot(v1, v1);
	float3 tv2 = v2 / dot(v2, v2);
	float3 u = tv1,v = tv2,w = -ray.direction;
	float l2w_array[16] = {
		u.x,v.x,w.x,0,
		u.y,v.y,w.y,0,
		u.z,v.z,w.z,0,
		0  ,0  ,0  ,1};

	Matrix<4,4> L2W(l2w_array);
	Matrix<4,4> W2L = L2W.inverse();
	

	float4 W = make_float4(ray.origin - anchor,1); 
	float4 L = W2L * W;
	float t = L.z;
	if(L.x<1&&L.y<1&&L.x>0&&L.y>0)
	{
		if (rtPotentialIntersection(t)) {
			shading_normal = geometric_normal = n;
			texcoord = make_float3(L.x, L.y, 0);
			geometry_color = make_float4(1.0f);

			refine_and_offset_hitpoint(ray.origin + t * ray.direction, ray.direction,
				n, anchor,
				back_hit_point, front_hit_point);

			rtReportIntersection(0);
		}
	}
	
	
}

rtDeclareVariable(float3, r_v, , );
rtDeclareVariable(float3, r_u, , );
RT_PROGRAM void bounds(int, float result[6])
{
	// v1 and v2 are scaled by 1./length^2.  Rescale back to normal for the bounds computation.
	const float3 tv1 = v1 / dot(v1, v1);
	const float3 tv2 = v2 / dot(v2, v2);
	const float3 p00 = anchor;
	const float3 p01 = anchor + tv1;
	const float3 p10 = anchor + tv2;
	const float3 p11 = anchor + tv1 + tv2;
	const float  area = length(cross(tv1, tv2));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf(fminf(p00, p01), fminf(p10, p11)) - make_float3(1000.0f);
		aabb->m_max = fmaxf(fmaxf(p00, p01), fmaxf(p10, p11)) + make_float3(1000.0f);
	}
	else {
		aabb->invalidate();
	}
}